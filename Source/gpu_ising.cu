#include "hip/hip_runtime.h"
#include "./Headers/gpu_ising.cuh"
#include "./Headers/lattice.cuh"

//CUDA enabled random number generator
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <stdio.h> //For testing


//*****************************
//      Private Functions     *
//*****************************


/**
 * Looks down in a given dimension, with periodic conditions
 * @param  loc - Current location on the given dimension
 * @return     - Returns an integer
 */
__device__ int gpu_Ising::LookDown(int loc){

        if((loc - 1) < 0)
                return (*LatticeSize - 1);
        else
                return (loc - 1);
};


/**
 * Looks up in a given dimension, with periodic conditions
 * @param  loc - Current location on the given dimension
 * @return     - Returns an integer
 */
__device__ int gpu_Ising::LookUp(int loc){

        if( (loc + 1) >= *LatticeSize)
                return 0;
        else
                return (loc + 1);
};



/**
 * Populates the sublattice based on the major lattice
 */
__device__ void gpu_Ising::PopulateSubLattice(){

        //Fill the normal spots
        SubLattice[SubLocation(minorX, minorY, minorZ, minorT, *LatticeSize)]
                = Lattice[MajLocation(majorX, majorY, majorZ, majorT, *LatticeSize)];

        //Fill looking up in the T direction
        SubLattice[SubLocation(minorX, minorY, minorZ, minorT + 1, *LatticeSize)]
                = Lattice[MajLocation(majorX, majorY, majorZ, LookUp(majorT), *LatticeSize)];

        //Fill looking Down in the T direction
        SubLattice[SubLocation(minorX, minorY, minorZ, minorT - 1, *LatticeSize)]
                = Lattice[MajLocation(majorX, majorY, majorZ, LookDown(majorT), *LatticeSize)];

        __syncthreads();

        //Fill looking up in the X direction
        if(minorY == blockDim.y && minorZ == blockDim.z) {

                SubLattice[SubLocation(minorX + 1, minorY, minorZ, minorT, *LatticeSize)]
                        = Lattice[MajLocation(LookUp(majorX), majorY, majorZ, majorT, *LatticeSize)];

        }

        //Fill looking down in the X direction
        if(minorY == 1 && minorZ == 1) {

                SubLattice[SubLocation(minorX - 1, minorY, minorZ, minorT, *LatticeSize)]
                        = Lattice[MajLocation(LookDown(majorX), majorY, majorZ, majorT, *LatticeSize)];

        }

        __syncthreads();
        //Fill looking up in the Y direction
        if(minorX == blockDim.x && minorZ == blockDim.z) {

                SubLattice[SubLocation(minorX, minorY + 1, minorZ, minorT, *LatticeSize)]
                        = Lattice[MajLocation(majorX, LookUp(majorY), majorZ, majorT, *LatticeSize)];

        }

        //Fill looking down in the Y direction
        if(minorX == 1 && minorZ == 1) {

                SubLattice[SubLocation(minorX, minorY + 1, minorZ, minorT, *LatticeSize)]
                        = Lattice[MajLocation(majorX, LookDown(majorY), majorZ, majorT, *LatticeSize)];

        }

        __syncthreads();

        //Fill looking up in the Z direction
        if(minorX == blockDim.x && minorY == blockDim.y) {

                SubLattice[SubLocation(minorX, minorY, minorZ + 1, minorT, *LatticeSize)]
                        = Lattice[MajLocation(majorX, majorY, LookUp(majorZ), majorT, *LatticeSize)];

        }

        //Fill looking down in the Z direction
        if(minorX == blockDim.x && minorY == blockDim.y) {

                SubLattice[SubLocation(minorX, minorY, minorZ - 1, minorT, *LatticeSize)]
                        = Lattice[MajLocation(majorX, majorY, LookDown(majorZ), majorT, *LatticeSize)];

        }

        __syncthreads();

};


/**
 * Gets the difference in energy based on neighboring spins
 * @param  old_spin - The old spin at the lattice site
 * @param  new_spin - The new spin at the lattice site
 * @return          - The energy difference
 */
__device__ double gpu_Ising::EnergyDiff(int old_spin, int new_spin){

        double sumNeighborSpin{0};

        //Look at neighbors in the X direction
        sumNeighborSpin += SubLattice[SubLocation(minorX + 1, minorY, minorZ, minorT, *LatticeSize)];
        sumNeighborSpin += SubLattice[SubLocation(minorX - 1, minorY, minorZ, minorT, *LatticeSize)];

        //Look at neighbors in the Y direction
        sumNeighborSpin += SubLattice[SubLocation(minorX, minorY + 1, minorZ, minorT, *LatticeSize)];
        sumNeighborSpin += SubLattice[SubLocation(minorX, minorY - 1, minorZ, minorT, *LatticeSize)];

        //Look at neighbors in the Z direction
        sumNeighborSpin += SubLattice[SubLocation(minorX, minorY, minorZ + 1, minorT, *LatticeSize)];
        sumNeighborSpin += SubLattice[SubLocation(minorX, minorY, minorZ - 1, minorT, *LatticeSize)];

        //Look at neighbors in the T direction
        sumNeighborSpin += SubLattice[SubLocation(minorX, minorY, minorZ, minorT + 1, *LatticeSize)];
        sumNeighborSpin += SubLattice[SubLocation(minorX, minorY, minorZ, minorT - 1, *LatticeSize)];

        return ((-1)*(*j)*sumNeighborSpin*(old_spin - new_spin)
                + (-1)*(*h)*(old_spin - new_spin));

};




//Returns the Boltzmann distribution of a given energy difference
__device__ double gpu_Ising::BoltzmannDist(double energydiff){
        return expf((-1)*(*beta)*energydiff);
};


//Equilibrate the 3D segments of the Lattice
__device__ void gpu_Ising::ThreeDEquilibrate(){

        int old_spin = SubLattice[SubLocation(minorX, minorY, minorZ, minorT, *LatticeSize)];
        int new_spin = (-1)*old_spin;
        double energydiff{0};

        int remainder = (minorX + minorY + minorZ)%2;

        int tid = MajLocation(threadIdx.x, (threadIdx.y + blockIdx.x * blockDim.y),
                              (threadIdx.z + blockIdx.y * blockDim.z), blockIdx.z, *LatticeSize);


        hiprandState_t rng;
        hiprand_init(clock64(), tid, 0, &rng);

        //Even 3D threads
        if(remainder == 0) {
                energydiff = EnergyDiff(old_spin, new_spin);

                //If the energy difference is lower or based on a
                //random probability accept the new spin.
                if(energydiff <= 0) {
                        SubLattice[SubLocation(minorX, minorY, minorZ, minorT, *LatticeSize)] = new_spin;
                } else if(hiprand_uniform(&rng) < BoltzmannDist(energydiff)) {
                        SubLattice[SubLocation(minorX, minorY, minorZ, minorT, *LatticeSize)] = new_spin;
                }
        }
        __syncthreads();



        //Odd 3D threads
        if(remainder == 1) {
                energydiff = EnergyDiff(old_spin, new_spin);

                //If the energy difference is lower or based on a
                //random probability accept the new spin.
                if(energydiff <= 0) {
                        SubLattice[SubLocation(minorX, minorY, minorZ, minorT, *LatticeSize)] = new_spin;
                } else if(hiprand_uniform(&rng) < BoltzmannDist(energydiff)) {
                        SubLattice[SubLocation(minorX, minorY, minorZ, minorT, *LatticeSize)] = new_spin;
                }
        }
        __syncthreads();

};




//****************************
//      Public Functions     *
//****************************





/**
 * Constructor for the GPU Ising Model object
 * @param  setConstants - Memory location of the lattice constants
 * @param  setLattice   - Memory location of the major lattice
 * @param  sharedlatt   - Memory location of the per-block(shared) lattice
 * @param  size       - Size of the sub lattice
 */
__device__ gpu_Ising::gpu_Ising(int *size, double *setbeta,
                                double *setj, double *seth,
                                int *SetLattice, int *SetSubLatt)
{
        LatticeSize = size;
        beta = setbeta;
        j = setj;
        h = seth;

        Lattice = SetLattice;
        SubLattice = SetSubLatt;

        //Find the location on the shared memory lattice
        minorX = threadIdx.x + 1;
        minorY = threadIdx.y + 1;
        minorZ = threadIdx.z + 1;
        minorT = 1;


        //Find the thread location on the major lattice
        majorX = threadIdx.x;
        majorY = threadIdx.y + blockIdx.x * blockDim.y;
        majorZ = threadIdx.z + blockIdx.y * blockDim.z;
        majorT = blockIdx.z;
};


/**
 * Each thread equilibrates the lattice
 */
__device__ void gpu_Ising::Equilibrate(){

        PopulateSubLattice();

        //Checkerboard pattern for 4D (ie odd/even T locations equilibrate)
        int remainder = blockIdx.z%2;

        //Even T dimension locations
        if(remainder == 0) {
                ThreeDEquilibrate();
        }
        __syncthreads();

        //Odd T dimension locations
        if(remainder == 1) {
                ThreeDEquilibrate();
        }
        __syncthreads();

        //Fill the normal spots
        Lattice[MajLocation(majorX, majorY, majorZ, majorT, *LatticeSize)]
                = SubLattice[SubLocation(minorX, minorY, minorZ, minorT, *LatticeSize)];

};
